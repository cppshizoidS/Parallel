
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <cstdio>
#include <string>
#include <chrono>
#include <vector>
#include <functional>

using namespace std;

struct comp_result
{
    int    value;
    string title;
    float  time;
};

using hrc       = chrono::high_resolution_clock;
using fdur      = chrono::duration<float>;
using fn_solver = function<int(string, string)>;
using fn_bench  = function<comp_result()>;

__device__ int d_count;

const int THREADS_PER_BLOCK = 64;

void
save_char_position(string& str, vector<vector<int>>& pos, char c, int charPos)
{
    if (pos.empty()) {
        pos = vector<vector<int>>(str.size());
    }

    for (int i = 0; i < str.size(); i++) {
        if (c == str[i]) {
            pos[i].push_back(charPos);
        }
    }
}

void
count_substr(
    vector<vector<int>>& pos,
    int* count,
    int prev_char_pos = -1,
    int cur_char      = 0
) {
    if (cur_char == pos.size()) {
        (*count)++;
        return;
    }

    for (auto& p : pos[cur_char]) {
        if (prev_char_pos < p) {
            count_substr(pos, count, p, cur_char + 1);
        }
    }
}

__device__ bool
check_combination(int* pos, int* sizes, int arraySize, long long combination)
{
    int prev_char_pos = -1;
    int sizesSum      = 0;

    for (int i = 0; i < arraySize; i++) {
        long long p = combination % sizes[i];

        if (prev_char_pos > pos[sizesSum + p]) {
            return false;
        }

        combination   /= sizes[i];
        prev_char_pos  = pos[sizesSum + p];
        sizesSum      += sizes[i];
    }

    return true;
}

__global__ void
count_substr(
    int* pos,
    long long combinations,
    int* sizes,
    int array_size,
    int combinations_per_thread
) {
    __shared__ int block_combinations;
    __shared__ int total_block_combinations;

    int id = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (id == 0) {
        d_count = 0;
    }

    if (threadIdx.x == 0) {
        block_combinations       = 0;
        total_block_combinations = 0;
    }

    int thread_combinations = 0;

    long long start_combination = id * combinations_per_thread;
    if (start_combination >= combinations) {
        return;
    }

    for (
        long long i = start_combination;
        i < combinations && i < start_combination + combinations_per_thread;
        i++
    ) {
        if (check_combination(pos, sizes, array_size, i)) {
            thread_combinations++;
        }
    }

    atomicAdd(&block_combinations, thread_combinations);
    __syncthreads();

    if (atomicExch(&total_block_combinations, 1) == 0) {
        atomicAdd(&d_count, block_combinations);
    }
}

void
get_blocks_and_threads(long long elems_count, int* blocks, int* threads)
{
    *blocks  = (elems_count - 1) / THREADS_PER_BLOCK + 1;
    *threads = (elems_count < THREADS_PER_BLOCK
        ? elems_count : THREADS_PER_BLOCK);
}

int
solve_parallel(string src, string sub)
{
    int *d_pos, *d_sizes;
    int blocks, threads, count;
    int thread_combinations = 100;
    int max_blocks          = 4000;
    int total_size          = 0;
    int offset              = 0;
    long long combinations  = 1;

    vector<int> sizes;
    vector<vector<int>> pos;

    for (int i = 0; i < src.size(); i++) {
        save_char_position(sub, pos, src[i], i);
    }

    for (auto& p : pos) {
        if (p.empty()) {
            return 0;
        }
    }

    for (auto& p : pos) {
        sizes.push_back(p.size());
    }

    for (auto& p : pos) {
        total_size   += p.size();
        combinations *= p.size();
    }

    hipMalloc(&d_pos, total_size * sizeof(int));

    for (int i = 0; i < pos.size(); i++) {
        hipMemcpy(
            d_pos + offset,
            &pos[i].front(),
            pos[i].size() * sizeof(int),
            hipMemcpyHostToDevice
        );
        offset += pos[i].size();
    }

    int bytes = pos.size() * sizeof(int);
    hipMalloc(&d_sizes, bytes);
    hipMemcpy(d_sizes, &sizes.front(), bytes, hipMemcpyHostToDevice);
  
    get_blocks_and_threads(
        combinations / thread_combinations + 1, &blocks, &threads);
  
    while (blocks > max_blocks) {
        thread_combinations *= 10;
        get_blocks_and_threads(
            combinations / thread_combinations + 1, &blocks, &threads);
    }
  
    count_substr <<< blocks, threads >>> (
        d_pos, combinations, d_sizes, sizes.size(), thread_combinations);
    hipDeviceSynchronize();
  
    hipMemcpyFromSymbol(&count, HIP_SYMBOL(d_count), sizeof(int));
  
    hipFree(d_pos);
    hipFree(d_sizes);
    return count;
}

int
solve_sequential(string src, string sub)
{
    int count = 0;
    vector<vector<int>> pos;

    for (int i = 0; i < src.size(); i++) {
        save_char_position(sub, pos, src[i], i);
    }

    for (auto& p : pos) {
        if (p.empty()) {
            return 0;
        }
    }

    count_substr(pos, &count);
    return count;
}

void
read_str(string* dest)
{
    cout << "enter string: " << flush;
    cin >> *dest;
}

bool
is_valid(const string& src, const string& sub)
{ return src.size() >= sub.size(); }

void
print_result(const comp_result& result)
{
    cout
        << "[" << result.title << "]:\n"
        << "time:  " << result.time  << "s\n"
        << "count: " << result.value << endl;
}

fn_bench
get_solver_bench(
    const string title,
    const string src,
    const string sub,
    fn_solver solver
) {
    return [title, src, sub, solver]() -> comp_result {
        hrc::time_point start{hrc::now()};
        
        int count = solver(src, sub);

        fdur time = chrono::duration_cast<fdur>(hrc::now() - start);
        return comp_result{count, title, time.count()};
    };
}

int
main(int argc, char** argv)
{
    int command;
    string src, sub;
    fn_bench bench;

    for (;;) {
        cout <<
            "\n"
            "source string: '" << src << "'\n"
            "substring:     '" << sub << "'\n"
            "1. set source string\n"
            "2. set substring\n"
            "3. run sequential\n"
            "4. run parallel\n"
            "0. exit\n"
            "==> " << flush;
        cin >> command;

        switch (command) {
        case 1:
            read_str(&src);
            break;
        case 2:
            read_str(&sub);
            break;
        case 3: // sequential
        case 4: // parallel
            if (!is_valid(src, sub)) {
                cout <<
                    "error: source string length must be greater "
                    "than substring length" << flush;
                continue;
            }

            if (command == 3) {
                bench = get_solver_bench(
                    "sequential", src, sub, solve_sequential);
            } else {
                bench = get_solver_bench("parallel", src, sub, solve_parallel);
            }
            print_result(bench());
            break;
        case 0:
            cout << "exit…\n";
            return 0;
        default:
            cout << "wrong input!\n";
            continue;
        }
    }
}

